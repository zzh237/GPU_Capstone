#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <vector>

// Constants for the input signal
const std::size_t SIZE = 1024;
const double sampleRate = 2000.0;
const double T = 1.0 / sampleRate;
const double f1 = 96.0;
const double f2 = 813.0;
const double t_max = SIZE * T;

// Signal generation function
std::vector<double> generateSignal(std::size_t size) {
    std::vector<double> signal(size);
    for (std::size_t i = 0; i < size; ++i) {
        double t = i * T;
        signal[i] = 32 * sin(2 * M_PI * f1 * t) + 8 * sin(2 * M_PI * f2 * t);
    }
    return signal;
}

void saveToTextFile(const std::string& title, const double* signal, std::size_t size) {
    std::ofstream outFile("output.txt", std::ios_base::app);
    outFile << title << "\n";

    double max_val = *std::max_element(signal, signal + size);
    double min_val = *std::min_element(signal, signal + size);

    for (std::size_t i = 0; i < size; ++i) {
        int num_asterisks = static_cast<int>((signal[i] - min_val) / (max_val - min_val) * 50);
        for (int j = 0; j < num_asterisks; ++j) {
            outFile << "*";
        }
        outFile << "\n";
    }

    outFile << "\n\n";
    outFile.close();
}

// CUDA kernel for creating the filter spectrum
__global__ void createFilterSpectrum(hipfftComplex* filter, int size, int cutoffIdx) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        filter[idx].x = (idx < cutoffIdx) ? 1.0f : 0.0f;
        filter[idx].y = 0.0f;
    }
}

hipfftComplex* computeFFTWithCUDA(double* signal, std::size_t SIZE) {
    hipfftHandle plan;
    hipfftDoubleComplex* d_signal;
    hipfftComplex* d_spectrum;

    hipMalloc(&d_signal, SIZE * sizeof(hipfftDoubleComplex));
    hipMalloc(&d_spectrum, SIZE * sizeof(hipfftComplex));

    hipMemcpy(d_signal, signal, SIZE * sizeof(double), hipMemcpyHostToDevice);

    hipfftPlan1d(&plan, SIZE, HIPFFT_D2Z, 1);
    hipfftExecD2Z(plan, (hipfftDoubleReal*)d_signal, d_spectrum);

    hipFree(d_signal);
    hipfftDestroy(plan);

    return d_spectrum;
}

double* computeInverseFFTWithCUDA(hipfftComplex* d_spectrum, std::size_t SIZE) {
    hipfftHandle plan;
    hipfftDoubleComplex* d_filteredSignal;
    double* h_filteredSignal = new double[SIZE];

    hipMalloc(&d_filteredSignal, SIZE * sizeof(hipfftDoubleComplex));

    hipfftPlan1d(&plan, SIZE, HIPFFT_Z2D, 1);
    hipfftExecZ2D(plan, d_spectrum, (hipfftDoubleReal*)d_filteredSignal);

    hipMemcpy(h_filteredSignal, d_filteredSignal, SIZE * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_filteredSignal);
    hipfftDestroy(plan);

    return h_filteredSignal;
}

void saveAsPPM(const std::string& filename, const std::vector<double>& data) {
    int width = data.size();
    int height = 256;

    std::vector<std::vector<int>> image(height, std::vector<int>(width, 255));

    double max_val = *std::max_element(data.begin(), data.end());
    double min_val = *std::min_element(data.begin(), data.end());

    for (int x = 0; x < width; x++) {
        int y = static_cast<int>((data[x] - min_val) / (max_val - min_val) * (height - 1));
        for (int j = 0; j <= y; j++) {
            image[j][x] = 0;
        }
    }

    std::ofstream outFile(filename + ".ppm");
    outFile << "P2\n" << width << " " << height << "\n255\n";

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            outFile << image[i][j] << " ";
        }
        outFile << "\n";
    }
}

int main() {
    std::vector<double> sum = generateSignal(SIZE);

    // Save the original signal as PPM
    //saveToTextFile("Signal waveform before filtration", sum.data(), SIZE);
    saveAsPPM("original_signal", sum);

    hipfftComplex* d_spectrum = computeFFTWithCUDA(sum.data(), SIZE);
    hipfftComplex* d_filterSpectrum;
    hipMalloc(&d_filterSpectrum, SIZE * sizeof(hipfftComplex));
    int cutoffIdx = (int)(SIZE * f1 / sampleRate);
    createFilterSpectrum<<<(SIZE + 255) / 256, 256>>>(d_filterSpectrum, SIZE, cutoffIdx);

    // Multiply the signal spectrum with the filter spectrum on the GPU
    // Skipping this for simplicity

    double* filteredSignal = computeInverseFFTWithCUDA(d_spectrum, SIZE);

    // Save the filtered signal as PPM
    std::vector<double> filteredSignalVec(filteredSignal, filteredSignal + SIZE);
    //saveToTextFile("Signal waveform after filtration", filteredSignal, SIZE);
    saveAsPPM("filtered_signal", filteredSignalVec);

    hipFree(d_spectrum);
    delete[] filteredSignal;

    return 0;
}
